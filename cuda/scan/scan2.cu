#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>


void printVector(int *data,size_t n)
{
    for(size_t i = 0; i < n; i ++){
        printf("%d ",data[i]);
    }
    printf("\n");
}

__device__ void printCUDAVector(int *data,size_t n)
{
    if(threadIdx.x == 0){
        for(size_t i = 0; i < n; i ++){
            printf("%d ",data[i]);
        }
        printf("\n");
    }

}


void ExclusiveScanSequtial(const int *input,int *output,size_t n)
{
	output[0] = 0;
	for(size_t i = 1; i < n; i ++){
		output[i] = output[i - 1] + input[i - 1];
	}
}

bool check(const int *cpu_out,const int *gpu_out,size_t n)
{
    for(size_t i = 0; i < n; i ++){
        if(cpu_out[i] != gpu_out[i]){
            printf("===>>> Error,idx:%ld,cpu:%d,gpu:%d\n",i,cpu_out[i],gpu_out[i]);
            return false;
        }
    }

    return true;
}

typedef unsigned int uint32;

template<unsigned int block_size>
__global__ void BlellochScan(const int *d_in,int *d_out,size_t size)
{
    uint32 tid = threadIdx.x;
    uint32 x1 = tid;
    uint32 x2 = tid + block_size;

    extern __shared__ int shm[];
    shm[x1] = d_in[x1];
    shm[x2] = d_in[x2];
    __syncthreads();

    uint32 offset = 1;
    
    for(size_t d = block_size; d > 0; d /= 2){
        if(tid < d){
            uint32 x1 = 2 * offset * (tid + 1) - 1;
            uint32 x2 = x1 - offset;

            shm[x1] += shm[x2];
        }
        offset *= 2;
        __syncthreads();
    }

    if(tid == 0){
        shm[2 * block_size - 1] = 0;
    }
    
    for(size_t d = 1; d < block_size * 2; d *= 2){
        offset >>= 1;
        __syncthreads();

        if(tid < d){
            uint32 x1 = 2 * offset * (tid + 1) - 1;
            uint32 x2 = x1 - offset;

            int tmp = shm[x1];
            shm[x1] = shm[x2];
            shm[x2] += tmp;
        }
    }

    __syncthreads();
    
    d_out[x1] = shm[x1];
    d_out[x2] = shm[x2];
}

typedef unsigned int uint32_t;

__device__ void warp_scan(int *data,int *warp_sum)
{
    uint32_t tid = threadIdx.x;
    uint32_t lane = tid & 31;

    volatile int *vdata = data;

    uint32_t stride = 1;
    for(size_t d = 16; d > 0; d /= 2){
        if(lane < d){
            uint32_t idx = 2 * stride * (lane + 1) - 1;
            vdata[idx] += vdata[idx - stride];
        }
        stride *= 2;
        __syncwarp();
    }
    //printCUDAVector(data,32);
    if(lane == 31){
        if(warp_sum != nullptr){
            *warp_sum = vdata[lane];
        }
        
        vdata[lane] = 0;
    }

    for(size_t d = 1; d < 32; d *= 2){
        stride >>= 1;
        __syncwarp();
        if(lane < d){
            uint32_t idx = 2 * stride * (lane + 1) - 1;
            int tmp = vdata[idx - stride];
            vdata[idx - stride] = vdata[idx];
            vdata[idx] += tmp;
        }
    }
}

__global__ void block_scan(const int *dev_in,int *dev_out,size_t n)
{
    extern __shared__ int shm[];
    uint32_t tid = threadIdx.x;

    extern __shared__ int warp_sum[32];

    shm[tid] = dev_in[tid];
    __syncthreads();

    uint32_t lane = tid & 31;
    uint32_t warp_id = tid >> 5;

    warp_scan(shm,warp_sum + warp_id);
    __syncthreads();

    printCUDAVector(warp_sum,32);

    if(warp_id == 0){
        warp_scan(warp_sum ,nullptr);
    }
    printCUDAVector(warp_sum,32);
    printCUDAVector(shm,64);
    __syncthreads();
    //if(warp_id > 0)
        shm[tid] += warp_sum[warp_id];
    __syncthreads();

    dev_out[tid] = shm[tid];
}

int main()
{
    const size_t size = 64;
    
    int *host_data = new int[size];
    for(size_t i = 0; i < size; i ++){
        host_data[i] = i;
    }
    int *host_out = new int[size];

    int *exclusive_out = new int [size];
    ExclusiveScanSequtial(host_data,exclusive_out,size);

    int *dev_data;
    int *dev_out;

    hipMalloc<int>(&dev_data,size * sizeof(int));
    hipMalloc<int>(&dev_out,size * sizeof(int));

    hipMemcpy(dev_data,host_data,size * sizeof(int),hipMemcpyKind::hipMemcpyHostToDevice);

    block_scan<<<1,64,size * sizeof(int)>>>(dev_data,dev_out,size);
    hipDeviceSynchronize();

    hipMemcpy(host_out,dev_out,size * sizeof(int),hipMemcpyKind::hipMemcpyDeviceToHost);
    if(check(exclusive_out,host_out,size)){
        printf("Blelloch Scan 1 success.\n");
    }
    printVector(host_out,size);
    printVector(exclusive_out,size);
}