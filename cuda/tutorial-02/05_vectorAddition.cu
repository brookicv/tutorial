#include <iostream>
#include <hip/hip_runtime.h>


#define N 5
__global__ void gpuAdd(int *d_a,int *d_b,int *d_c)
{
    int tid = blockIdx.x; // Block index of current kernel
    if(tid < N){
        d_c[tid] = d_a[tid] + d_b[tid];
    }
    printf("Block idx:%d\n",tid);
}

int main()
{  
    int *d_a,*d_b,*d_c;
    // allocate the memory
    hipMalloc((void**)&d_a,N * sizeof(int));
    hipMalloc((void**)&d_b,N * sizeof(int));
    hipMalloc((void**)&d_c,N * sizeof(int));

    int h_a[N],h_b[N],h_c[N];
    for(int i = 0; i < N ; i ++) {
        h_a[i] = 2 * i * i;
        h_b[i] = i;
    }

    // copy data from host to device memory
    hipMemcpy(d_a,h_a,N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,N * sizeof(int),hipMemcpyHostToDevice);

    // kernel call
    gpuAdd<<<N,1>>>(d_a,d_b,d_c);

    hipMemcpy(h_c,d_c,N * sizeof(int),hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i ++) {
        printf("The sum of %d element is %d + %d = %d\n",i,h_a[i],h_b[i],h_c[i]);
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}