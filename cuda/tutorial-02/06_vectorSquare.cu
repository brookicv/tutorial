#include <iostream>
#include <hip/hip_runtime.h>


#define N 5

__global__ void gpuSquare(float *d_in,float *d_out)
{
    int tid = threadIdx.x;
    float temp = d_in[tid];
    d_out[tid] = temp * temp;
    printf("Thread idx:%d\n",tid);
}

int main()
{
    float *d_in,*d_out;
    hipMalloc((void **)&d_in,N * sizeof(float));
    hipMalloc((void **)&d_out,N * sizeof(float));

    float h_in[N];
    float h_out[N];
    for(int i = 0; i < N; i ++) {
        h_in[i] = i;
    }

    // copy data from host to device memory
    hipMemcpy(d_in,h_in, N * sizeof(float),hipMemcpyHostToDevice);

    // kernel call
    gpuSquare<<<1,N>>>(d_in,d_out);

    // copy result
    hipMemcpy(h_out,d_out, N * sizeof(float),hipMemcpyDeviceToHost);

    for(int i = 0 ;i < N ; i ++) {
        printf("The quare of %f is %f\n",h_in[i],h_out[i]);
    }

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}