#include "hip/hip_runtime.h"
#include "cudastart.h"
#include <hip/hip_runtime.h>
#include <chrono>


void sumMatrixOnCpu(float *matrixA,float *matrixB,float *matrixC,unsigned int nx,unsigned int ny)
{
    for(int i = 0; i < ny; i ++){

        // 指针移到第i行
        float *a = matrixA + i * nx;
        float *b = matrixB + i * nx;
        float *c = matrixC + i * nx;
        for(int j = 0; j < nx; j ++){
            c[j] = a[j] + b[j];
        }
    }
}

__global__ void sumMatrixOnGpu(float *matrixA,float *matrixB,float *matrixC,unsigned int nx,unsigned int ny)
{
    /*
        每个线程处理一个矩阵元素的加法
        线程分布在多个block中，利用下面公司计算出，当前线程要计算的矩阵元素的坐标
    */
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // 根据当前线程的坐标，计算出线程计算的矩阵元素的坐标
    unsigned int idx = ix + iy * nx; 

    // 防止访问越界，要加限定条件
    // 优化点，访问存储，每计算一个元素都要写回Globel Memory
    // 利用shared memory 缓存计算结果，分批次写回Globa Memory 
    // 但是，shared memory没有大空间
    // 使用寄存器缓存，限制每个block的线程个数
    if(ix < nx && iy < ny){
        matrixC[idx] = matrixA[idx] + matrixB[idx];
    }

    /*
        进行一次加法运算，需要从Global Memory中取两词数据，
        并且需要将结果写回Global Memory
        合并访问，由于一个warp中从Global Memory中连续取32个数，
        会进行合并访问主存，也就是一次计算，一个warp只需要访问
        2次Global Memory
        但是向Global Memory的写回操作，却没有办法合并。
    */

}

__global__ void sumMatrixOnGPURegister(float *matrixA,float *matrixB,float *matrixC,unsigned int nx,unsigned int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
}

int main()
{
    printf("hello cuda\n");
    initDevice(0);

    // 输入二维矩阵，4096*4096，float
    const unsigned int nx = 1 << 13;
    const unsigned int ny = 1 << 13;
    const unsigned int nBytes = nx * ny * sizeof(float);

    float *h_a = new float[nBytes];
    float *h_b = new float[nBytes];
    float *h_c = new float[nBytes];
    float *h_reslut = new float[nBytes];

    initialData(h_a,nx*ny);
    initialData(h_b,nx * ny);

    float *d_a,*d_b,*d_c;
    CHECK(hipMalloc((void **)&d_a,nBytes));
    CHECK(hipMalloc((void **)&d_b,nBytes));
    CHECK(hipMalloc((void **)&d_c,nBytes));

    CHECK(hipMemcpy(d_a,h_a,nBytes,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b,h_b,nBytes,hipMemcpyHostToDevice));

    // 线程块
    // 每一个线程计算一个位置的和
    // 32,一个wrap的线程个数，线程是以wrap为最小单元调度的
    dim3 block(32,32);
    dim3 grid((nx -1 ) / block.x + 1,(ny -1) / block.y + 1);

    printf("grid.x = %d,grid.y = %d\n",grid.x,grid.y);
    printf("All threads:%d\n",grid.x * grid.y * block.x * block.y);

    // 创建cuda event
    hipEvent_t event_start,stop;
    hipEventCreate(&event_start);
    hipEventCreate(&stop);
    // 测试cpu时间
    auto start = std::chrono::steady_clock::now();

    // kernel call
    hipEventRecord(event_start);
    
    sumMatrixOnGpu<<<grid,block>>>(d_a,d_b,d_c,nx,ny);

    hipEventRecord(stop);

    hipEventSynchronize(event_start); // 等待时间结束
    hipEventSynchronize(stop);

    float eventSpan;
    hipEventElapsedTime(&eventSpan,event_start,stop);

    CHECK(hipDeviceSynchronize());

    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::steady_clock::now() - start);
    auto ms_duration = duration.count() / 1000000.0f;
    printf("Gpu execution time:%lf ms\n",ms_duration);

    printf("Gpu envent time:%lf ms\n",eventSpan);

    // cpu
    hipMemcpy(h_reslut,d_c,nBytes,hipMemcpyDeviceToHost);

    start = std::chrono::steady_clock::now();

    sumMatrixOnCpu((float*)h_a,(float*)h_b,(float*)h_c,nx,ny);
    duration = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::steady_clock::now() - start);
    auto ms  = duration.count() / 1000000.0f;
    printf("Cpu execution time:%lf ms \n",ms);

    checkResult((float*)h_c,(float*)h_reslut,nx * ny);

    // free
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    delete[] h_reslut;

    return 0;
}