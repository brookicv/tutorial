#include <iostream>
#include <hip/hip_runtime.h>


#define N 50

__global__ void gpuAdd(int *d_a,int *d_b,int *d_c)
{
    // thread的唯一编号
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    printf("=>>Thread id:%d\n",tid);
    while(tid < N)
    {
        d_c[tid] = d_a[tid] + d_b[tid];
        tid += blockDim.x * gridDim.x;
        printf("====>>thread id:%d\n",tid);
    }
}

int main()
{
    int *d_a,*d_b,*d_c;
    hipMalloc((void **)&d_a,N * sizeof(int));
    hipMalloc((void **)&d_b,N * sizeof(int));
    hipMalloc((void **)&d_c,N * sizeof(int));

    int h_a[N],h_b[N],h_c[N];
    for(int i = 0; i < N; i ++) {
        h_a[i] = 2 * i * i;
        h_b[i] = i;
    }

    // copy data from host to device memory
    hipMemcpy(d_a,h_a,N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,N * sizeof(int),hipMemcpyHostToDevice);

    // kernel call
    gpuAdd<<<2,2>>>(d_a,d_b,d_c); 

    hipMemcpy(h_c,d_c,N * sizeof(int),hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    for(int i = 0 ;i < N; i ++) {
        printf("%d + %d = %d\n",h_a[i],h_b[i],h_c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}